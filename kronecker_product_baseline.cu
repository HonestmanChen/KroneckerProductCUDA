#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"
#include "string.h"
#include "time.h"

#define A_w 50
#define A_h 50
#define B_w 32
#define B_h 32

typedef struct{
	int width;
	int height;
	float * elements;
}Matrix;

// #define 

void rightKronecker1(Matrix A, Matrix B, Matrix C){
	for(int c_row=0; c_row<C.height; c_row++){
		for(int c_col=0; c_col<C.width; c_col++){
			C.elements[c_col + c_row*C.width] = 
			A.elements[c_col/B.width + c_row/B.height * A.width]
			* B.elements[c_col%B.width + c_row%B.height*B.width];
		}
	}
}

void rightKronecker2(Matrix A, Matrix B, Matrix C){
	for(int a_row=0; a_row<A.height; a_row++){
		for(int a_col=0; a_col<A.width; a_col++){
			for(int b_row=0; b_row<B.height; b_row++){
				for(int b_col=0; b_col<B.width; b_col++){
					C.elements[(b_col+a_col*B.width)+(b_row+a_row*B.height)*A.width*B.width] 
					= A.elements[a_col+a_row*A.width] * B.elements[b_col+b_row*B.width];
				}
			}
		}
	}
}


void generatorNum(float* array, int num)
{
//	srand((unsigned)time(NULL));
	for(int i=0;i<num;i++)
	{
		array[i]=rand()%5;
	}
}

void printUsage(void)
{
                printf("\n");
                printf("The program aims to calculate the product of matrix A and B\n");
                printf("-h matrix A row num\n");
                printf("-w matrix A col num\n");
                printf("-H matrix B row num\n");
                printf("-W matrix B col num\n");
}

int main(int argc,char** argv){

	// int A_w,B_w,A_h,B_h;
 //    if(argc==1)
 //    {
 //        printf("Error: no enough parameters.Please input the col and row number of Matrix A and B,respectively\n");
 //        exit(0);
 //    }
 //    else if(argc==2)
 //    {
 //        if(strcmp("--help",argv[1])==0)
 //        {
 //            printUsage();
 //            exit(0);
 //        }

 //    }
	// for(int id=1;id<argc;id+=2)
 //    {
 //        if(strcmp("-h",argv[id])==0)
 //                A_h=atoi(argv[id+1]);
 //        else if(strcmp("-w",argv[id])==0)
 //                A_w=atoi(argv[id+1]);
 //        else if(strcmp("-W",argv[id])==0)
 //                B_w=atoi(argv[id+1]);
 //        else if(strcmp("-H",argv[id])==0)
 //                B_h=atoi(argv[id+1]);
 //    }
    

    // Matrix A,d_A,B,d_B,C,d_C;
    Matrix A, B, C1, C2;
    A.width=A_w;A.height=A_h;
    B.width=B_w;B.height=B_h;
    C1.width=A_w*B_w;C1.height=A_h*B_h;
    C2.width=A_w*B_w;C2.height=A_h*B_h;

    A.elements=(float *)malloc(A.width*A.height*sizeof(float));
	B.elements=(float *)malloc(B.width*B.height*sizeof(float));
	C1.elements=(float *)malloc(C1.width*C1.height*sizeof(float));
	C2.elements=(float *)malloc(C2.width*C2.height*sizeof(float));

 //    A.elements=(float *)malloc(A.width*A.height*sizeof(float));
	// B.elements=(float *)malloc(B.width*B.height*sizeof(float));
	// C.elements=(float *)malloc(C.width*C.height*sizeof(float));

   	generatorNum(A.elements,A.width*A.height);
	generatorNum(B.elements,B.width*B.height);
	memset(C1.elements,0,C1.width*sizeof(float)*C1.height);
	memset(C2.elements,0,C2.width*sizeof(float)*C2.height);

	// printf("A.elements:\n");
	// for(int i=0;i<A.height;i++){
	// 	for(int j=0;j<A.width;j++){
	// 		printf("%d ", int(A.elements[j+i*A.width]));
	// 	}
	// printf("\n");
	// }
	// printf("B.elements:\n");
	// for(int i=0;i<B.height;i++){
	// 	for(int j=0;j<B.width;j++){
	// 		printf("%d ", int(B.elements[j+i*B.width]));
	// 	}
	// printf("\n");
	// }

	srand(time(0));
	clock_t start,finish1, finish2;
	start=clock();
	rightKronecker1(A, B, C1);
	finish1=clock();
	rightKronecker2(A, B, C2);
	finish2=clock();

	// printf("C1.elements:\n");
	// for(int i=0;i<C1.height;i++){
	// 	for(int j=0;j<C1.width;j++){
	// 		printf("%d ", C1.elements[j+i*C1.width]);
	// 	}
	// printf("\n");
	// }

	// printf("C2.elements:\n");
	// for(int i=0;i<C2.height;i++){
	// 	for(int j=0;j<C2.width;j++){
	// 		printf("%d ", C2.elements[j+i*C2.width]);
	// 	}
	// printf("\n");
	// }

	printf("Difference between 2 method:\n");
	float diff = 0;
	for(int i=0;i<C2.height;i++){
		for(int j=0;j<C2.width;j++){
			diff = C2.elements[j+i*C2.width] - C1.elements[j+i*C2.width];
		}
	}
	printf("%f\n", diff);

	printf("method1 cost time %f ms\n",(finish1-start)*1000.0/CLOCKS_PER_SEC);
	printf("method2 cost time %f ms\n",(finish2-finish1)*1000.0/CLOCKS_PER_SEC);
	// malloc matrix A B C on GPU
	// cudaMalloc(&d_A.elements,sizeof(float)*A.width*A.height);
	// cudaMalloc(&d_B.elements,sizeof(float)*B.width*B.height);
	// cudaMalloc(&d_C.elements,sizeof(float)*C.width*C.height);

	return 0;


}