#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"
#include "string.h"
#include "time.h"
#include "math.h"

#define block_size 1024
#define A_w 50
#define A_h 50
#define B_w 32
#define B_h 32

typedef struct{
	int width;
	int height;
	float * elements;
}Matrix;



__global__ void rightKronecker_gpu2(float *A, float *B, float *C, 
	int A_height, int A_width, int B_height, int B_width, int C_height, int C_width){
	
	int bid = (blockIdx.x );
	int a_col = bid%A_width ;
	int a_row = bid/A_width;

	int tid = (threadIdx.x  );//
	int b_col = tid%B_width ;
	int b_row = tid/B_width;

	if(bid<A_width*A_height&&tid<B_width*B_height){
		
		C[(b_col+a_col*B_width)+(b_row+a_row*B_height)*A_width*B_width] = 
			A[a_col + a_row * A_width]
			* B[b_col + b_row * B_width];
		
		
		// tid += blockIdx.x*blockDim.x;
		__syncthreads();
		
		
	}
}

void rightKronecker_cpu1(Matrix A, Matrix B, Matrix C){
	for(int c_row=0; c_row<C.height; c_row++){
		for(int c_col=0; c_col<C.width; c_col++){
			C.elements[c_col + c_row*C.width] = 
			A.elements[c_col/B.width + c_row/B.height * A.width]
			* B.elements[c_col%B.width + c_row%B.height*B.width];
		}
	}
}

void rightKronecker_cpu2(Matrix A, Matrix B, Matrix C){
	for(int a_row=0; a_row<A.height; a_row++){
		for(int a_col=0; a_col<A.width; a_col++){
			for(int b_row=0; b_row<B.height; b_row++){
				for(int b_col=0; b_col<B.width; b_col++){
					C.elements[(b_col+a_col*B.width)+(b_row+a_row*B.height)*A.width*B.width] 
					= A.elements[a_col+a_row*A.width] * B.elements[b_col+b_row*B.width];
				}
			}
		}
	}
}

void generatorNum(float* array, int num)
{

	for(int i=0;i<num;i++)
	{
		array[i]=rand()%5;
	}
}

void printUsage(void)
{
    printf("\n");
    printf("The program aims to calculate the product of matrix A and B\n");
    printf("-h matrix A row num\n");
    printf("-w matrix A col num\n");
    printf("-H matrix B row num\n");
    printf("-W matrix B col num\n");
}

void displayMatrx(Matrix X){
	printf(".elements:\n");
	for(int i=0;i<X.height;i++){
		for(int j=0;j<X.width;j++){
			printf("%.1f ", X.elements[j+i*X.width]);
		}
	printf("\n");
	}
}

void computeDiff(Matrix X1, Matrix X2){
	float diff = 0;
	if(X1.height==X2.height && X1.width==X2.width){
		for(int i=0;i<X2.height;i++){
			for(int j=0;j<X2.width;j++){
				diff += abs(X2.elements[j+i*X2.width] - X1.elements[j+i*X1.width]);
			}
		}
		printf("%f\n", diff);
	}
}


int main(int argc,char** argv){
// if use command to get matrix size
	// int A_w,B_w,A_h,B_h;
 //    if(argc==1)
 //    {
 //        printf("Error: no enough parameters.Please input the col and row number of Matrix A and B,respectively\n");
 //        exit(0);
 //    }
 //    else if(argc==2)
 //    {
 //        if(strcmp("--help",argv[1])==0)
 //        {
 //            printUsage();
 //            exit(0);
 //        }

 //    }
	// for(int id=1;id<argc;id+=2)
 //    {
 //        if(strcmp("-ah",argv[id])==0)
 //                A_h=atoi(argv[id+1]);
 //        else if(strcmp("-aw",argv[id])==0)
 //                A_w=atoi(argv[id+1]);
 //        else if(strcmp("-bw",argv[id])==0)
 //                B_w=atoi(argv[id+1]);
 //        else if(strcmp("-bh",argv[id])==0)
 //                B_h=atoi(argv[id+1]);
 //    }
    // A_h = 50;
    // A_w = 50;
    // B_h = 50;
    // B_w = 50;
    

    // Matrix A,d_A,B,d_B,C,d_C;
    Matrix A, B, C_cpu1, C_cpu2, C_gpu;
    float *d_A, *d_B, *d_C;
    A.width=A_w;
    A.height=A_h;
    B.width=B_w;
    B.height=B_h;
    
    C_cpu1.width=A_w*B_w;
    C_cpu1.height=A_h*B_h;
    C_cpu2.width=A_w*B_w;
    C_cpu2.height=A_h*B_h;
    C_gpu.width=A_w*B_w;
    C_gpu.height=A_h*B_h;
   

    A.elements=(float *)malloc(A.width*A.height*sizeof(float));
	B.elements=(float *)malloc(B.width*B.height*sizeof(float));
	C_cpu1.elements=(float *)malloc(C_cpu1.width*C_cpu1.height*sizeof(float));
	C_cpu2.elements=(float *)malloc(C_cpu2.width*C_cpu2.height*sizeof(float));
	C_gpu.elements=(float *)malloc(C_gpu.width*C_gpu.height*sizeof(float));
	


   	generatorNum(A.elements,A.width*A.height);
	generatorNum(B.elements,B.width*B.height);
	memset(C_cpu1.elements,0,C_cpu1.width*sizeof(float)*C_cpu1.height);
	memset(C_cpu2.elements,0,C_cpu2.width*sizeof(float)*C_cpu2.height);
	memset(C_gpu.elements,0,C_gpu.width*sizeof(float)*C_gpu.height);


	
	hipMalloc(&d_A,sizeof(float)*A.width*A.height);
	hipMalloc(&d_B,sizeof(float)*B.width*B.height);
	hipMalloc(&d_C,sizeof(float)*C_gpu.width*C_gpu.height);
	
	
	// dim3 block(block_size,block_size);
	// dim3 grid((C3.width-1+block_size)/block_size,(C3.height-1+block_size)/block_size);
	
	dim3 block(block_size);
	dim3 grid((C_gpu.width-1+block_size)/block_size);

	srand(time(0));
	clock_t start_cpu1,start_cpu2, start_gpu, start_gpu_pure, 
	finish_cpu1, finish_cpu2, finish_gpu, finish_gpu_pure;
	
	//cpu1
	start_cpu1=clock();
	rightKronecker_cpu1(A, B, C_cpu1);
	finish_cpu1=clock();
	//cpu2
	start_cpu2=clock();
	rightKronecker_cpu2(A, B, C_cpu2);
	finish_cpu2=clock();

	//gpu1**********************************************************
	start_gpu = clock();
	hipMemcpy(d_A,A.elements,A.width*A.height*sizeof(float),hipMemcpyHostToDevice);	
	hipMemcpy(d_B,B.elements,B.width*B.height*sizeof(float),hipMemcpyHostToDevice);
	
	start_gpu_pure = clock();

	rightKronecker_gpu2<<<A.width*A.height, block_size>>>
	(d_A, d_B, d_C, A.height, A.width, B.height, B.width, C_gpu.height, C_gpu.width);
	
	finish_gpu_pure = clock();
	
	hipMemcpy(C_gpu.elements,d_C,C_gpu.width*C_gpu.height*sizeof(float),hipMemcpyDeviceToHost);


	finish_gpu = clock();

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);


	

	printf("Difference between cpu1 and cpu2:\t");
	computeDiff(C_cpu1, C_cpu2);
	printf("Difference between cpu1 and gpu2:\t");
	computeDiff(C_cpu1, C_gpu);
	
	printf("cpu1 cost time %f ms\n",(finish_cpu1 - start_cpu1)*1000.0/CLOCKS_PER_SEC);
	printf("cpu2 cost time %f ms\n",(finish_cpu2 - start_cpu2)*1000.0/CLOCKS_PER_SEC);
	printf("gpu2 cost time %f ms\tpure computing %f ms\n",(finish_gpu - start_gpu)*1000.0/CLOCKS_PER_SEC, (finish_gpu_pure - start_gpu_pure)*1000.0/CLOCKS_PER_SEC);


	

	free(A.elements);
	free(B.elements);
	free(C_cpu1.elements);
	free(C_cpu2.elements);
	free(C_gpu.elements);
	


	return 0;


}